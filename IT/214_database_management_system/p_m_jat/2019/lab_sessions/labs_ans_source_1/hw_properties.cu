#include<stdio.h>
#include<hip/hip_runtime.h>

int main()

{

    int devCount;

    hipGetDeviceCount(&devCount);

    printf("CUDA Device Query...\n");

    printf("There are %d CUDA devices.\n", devCount);

 
    for (int i = 0; i < devCount; ++i)
    {

        // Get device properties

        printf("\nCUDA Device #%d\n", i);

        hipDeviceProp_t devProp;

        hipGetDeviceProperties(&devProp, i);

        printf("Device Name: %s\n", devProp.name);
        printf("Total Global Memory: %d\n", devProp.totalGlobalMem);
		printf("Maximum Threads per Block: %d\n", devProp.maxThreadsPerBlock);
		printf("Maximum Threads Dimension in X-axis: %d\n", devProp.maxThreadsDim[0]);
		printf("Maximum Threads Dimension in Y-axis: %d\n", devProp.maxThreadsDim[1]);
		printf("Maximum Threads Dimension in Z-axis: %d\n", devProp.maxThreadsDim[2]);
		printf("Maximum Grid Size in X-axis: %d\n", devProp.maxGridSize[0]);
		printf("Maximum Grid Size in Y-axis: %d\n", devProp.maxGridSize[1]);
		printf("Maximum Grid Size in Z-axis: %d\n", devProp.maxGridSize[2]);
		printf("Warp Size: %d\n", devProp.warpSize);
		printf("Clock Rate: %d\n", devProp.clockRate);
	printf("Shared Memory Per Block: %d\n", devProp.sharedMemPerBlock);
	printf("Registers Per Block: %d\n", devProp.regsPerBlock);
	//printf("");
    }

    return 0;

}
