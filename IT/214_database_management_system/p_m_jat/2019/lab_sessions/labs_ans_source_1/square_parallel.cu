#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
 
// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(double *a, double *c, long long n)
{
	// Get our global thread ID
    	int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    	// Make sure we do not go out of bounds
    	if (id < n)
        	c[id] = a[id]*a[id];
}
 
int main( int argc, char* argv[] )
{
    	FILE *fptr = fopen("parallel_square.txt", "w");
	
	long long minsize = pow(2,8);
	long long maxsize = pow(2,28);
	//int cnt = 0;
	//int  n = 20;
	
	long long n;
	
	for(n = minsize; n<maxsize; n*=2)
	{

	// Size of vectors
    	//int n = 1000000;
 
    	// Host input vectors
    	double *h_a;
    	//double *h_b;
    	//Host output vector
    	double *h_c;
 
    	// Device input vectors
    	double *d_a;
    	//double *d_b;
    	//Device output vector
    	double *d_c;
 
    	// Size, in bytes, of each vector
    	size_t bytes = n*sizeof(double);
 
    	// Allocate memory for each vector on host
    	h_a = (double*)malloc(bytes);
    	//h_b = (double*)malloc(bytes);
    	h_c = (double*)malloc(bytes);
 
    	// Allocate memory for each vector on GPU
    	hipMalloc(&d_a, bytes);
    	//cudaMalloc(&d_b, bytes);
    	hipMalloc(&d_c, bytes);
 
    	long long i;
    	// Initialize vectors on host
    	for( i = 0; i < n; i++ ) {
        	h_a[i] = sin(i);
        	//h_b[i] = cos(i);
    	}
 
    	hipEvent_t start, stop;
    	hipEventCreate(&start);
    	hipEventCreate(&stop);		

    	// Copy host vectors to device
    	hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    	//cudaMemcpy( d_b, h_b, bytes, cudaMemcpyHostToDevice);
 
    	int blockSize, gridSize;
 
    	// Number of threads in each thread block
    	blockSize = 1024;
 
    	// Number of thread blocks in grid
    	gridSize = (int)ceil((float)n/blockSize);
 
    	hipEventRecord(start);		
    	// Execute the kernel
    	vecAdd<<<gridSize, blockSize>>>(d_a, d_c, n);
 	hipEventRecord(stop);

	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

    	// Copy array back to host
    	hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
 
    	// Sum up vector c and print result divided by n, this should equal 1 within error
   	// double sum = 0;
   	// for(i=0; i<n; i++)
   	//     sum += h_c[i];
   	// printf("final result: %f\n", sum/n);
 
    	// Release device memory
    	hipFree(d_a);
    	//cudaFree(d_b);
    	hipFree(d_c);
 
	fprintf(fptr, "%ld %lf\n", n, milliseconds);
//	printf("%ld %lf\n", n, milliseconds);
    	// Release host memory
    	free(h_a);
    	//free(h_b);
    	free(h_c);
 	}
	fclose(fptr);
    	return 0;
}
